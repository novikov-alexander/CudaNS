#include "hip/hip_runtime.h"
#include "header.hpp"
#include <algorithm>

//---------------------------------------------------------------------
// this function performs the solution of the approximate factorization
// step in the y-direction for all five matrix components
// simultaneously. The Thomas algorithm is employed to solve the
// systems for the y-lines. Boundary conditions are non-periodic
//---------------------------------------------------------------------

void y_solve_two(
    dim3 blocks, dim3 threads,
    dim3 blocks2, dim3 threads2,
    double *lhs_, double *lhsp_, double *lhsm_, double *rhs, double *rho_i, double *vs, double *speed, double c3c4, double dy3, double con43, double dy5, double c1c5, double dy1, double dtty2, double dtty1, double dymax, double c2dtty1, double comz1, double comz4, double comz5, double comz6,
    int nx2, int ny2, int nz2, int ny)
{
    run_solve_kernels(blocks, threads, blocks2, threads2, (double *)lhs_, (double *)lhsp_, (double *)lhsm_, rhs, (double *)rho_i, (double *)us, (double *)speed, c3c4, dy3, con43, dy5, c1c5, dy1, dtty2, dtty1, dymax, c2dtty1, comz1, comz4, comz5, comz6, nz2, nx2, ny2, ny);
}

void y_solve_inversion(dim3 blocks, dim3 threads, double *rhs, double bt, int nx2, int ny2, int nz2)
{
    run_inversion_kernels(blocks, threads, rhs, bt, nz2, nx2, ny2);
}

#define src(x, y, z) src[z + (y) * P_SIZE + (x) * P_SIZE * P_SIZE]
#define dst(x, y, z) dst[x + (z) * P_SIZE + (y) * P_SIZE * P_SIZE]
__global__ void y_solve_transpose_3D(double *dst, double *src, int nx2, int ny2, int nz2)
{
    int m;

    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int i = threadIdx.z + blockIdx.z * blockDim.z;

    if ((k <= nz2 + 1) && (j <= ny2 + 1) && (i <= nx2 + 1))
    {
        dst(i, j, k) = src(i, j, k);
    }
}

__global__ void y_solve_inv_transpose_3D(double *dst, double *src, int nx2, int ny2, int nz2)
{
    int m;

    int k = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int i = threadIdx.z + blockIdx.z * blockDim.z;

    if ((k <= nz2 + 1) && (j <= ny2 + 1) && (i <= nx2 + 1))
    {
        src(i, j, k) = dst(i, j, k);
    }
}

#undef src
#undef dst

void y_solve()
{
    int i, j, k, j1, j2, m;
    double ru1, rhoq1, fac1, fac2;

    const int size5 = sizeof(double) * P_SIZE * P_SIZE * P_SIZE * 5;
    const int size = sizeof(double) * P_SIZE * P_SIZE * P_SIZE;

    dim3 blocks = dim3(nx2 / 32 + 1, ny2, nz2);
    dim3 threads = dim3(32, 1, 1);

    dim3 blocks2 = dim3(nx2 / 32 + 1, nz2 / 8 + 1);
    dim3 threads2 = dim3(32, 8);

    dim3 blockst = dim3(nx / 8 + 1, ny / 8 + 1, nz / 8 + 1);
    dim3 threadst = dim3(8, 8, 8);

    if (timeron)
        timer_start(t_ysolve);

    y_solve_transpose_3D<<<blockst, threadst>>>((double *)gpuTmp3D, (double *)gpuVs, nx2, ny2, nz2);
    std::swap(gpuTmp3D, gpuVs);
    hipDeviceSynchronize();

    y_solve_two(
        blocks, threads,
        blocks2, threads2,
        (double *)lhs_gpu, (double *)lhsp_gpu, (double *)lhsm_gpu, (double *)gpuRhs, (double *)gpuRho_i, (double *)gpuVs, (double *)gpuSpeed, c3c4, dy3, con43, dy5, c1c5, dy1, dtty2, dtty1, dymax, c2dtty1, comz1, comz4, comz5, comz6, nx2, ny2, nz2, ny);

    //---------------------------------------------------------------------
    // block-diagonal matrix-vector multiplication
    //---------------------------------------------------------------------
    if (timeron)
        timer_start(t_pinvr);

    y_solve_inversion(blocks, threads, (double *)gpuRhs, bt, nx2, ny2, nz2);

    if (timeron)
        timer_stop(t_pinvr);

    std::swap(gpuTmp3D, gpuRho_i);
    y_solve_inv_transpose_3D<<<blockst, threadst>>>((double *)gpuTmp3D, (double *)gpuRho_i, nx2, ny2, nz2);
    std::swap(gpuTmp3D, gpuVs);
    y_solve_inv_transpose_3D<<<blockst, threadst>>>((double *)gpuTmp3D, (double *)gpuVs, nx2, ny2, nz2);
    std::swap(gpuTmp3D, gpuSpeed);
    y_solve_inv_transpose_3D<<<blockst, threadst>>>((double *)gpuTmp3D, (double *)gpuSpeed, nx2, ny2, nz2);

    if (timeron)
        timer_stop(t_ysolve);
}

#undef lhs
