#include "hip/hip_runtime.h"
#include "header.hpp"

__global__ void solve_kernel_one(double *lhs_, double *lhsp_, double *lhsm_, int nx2, int ny2, int nz2)
{
    int m;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;

    // part 1
    if (i <= nx2 && j <= ny2)
    {
#pragma unroll 5
        for (m = 0; m < 5; m++)
        {
            lhs_(i, j, 0, m) = lhs_(i, j, nz2 + 1, m) = 0.0;
            lhsp_(i, j, 0, m) = lhsp_(i, j, nz2 + 1, m) = 0.0;
            lhsm_(i, j, 0, m) = lhsm_(i, j, nz2 + 1, m) = 0.0;
        }

        lhs_(i, j, 0, 2) = lhs_(i, j, nz2 + 1, 2) = 1.0;
        lhsp_(i, j, 0, 2) = lhsp_(i, j, nz2 + 1, 2) = 1.0;
        lhsm_(i, j, 0, 2) = lhsm_(i, j, nz2 + 1, 2) = 1.0;
    }
}

#undef us
#undef speed
#define us(x, y, z) us[INDEX_3D(y, z, x)]
#define speed(x, y, z) speed[INDEX_3D(y, z, x)]
__global__ void solve_kernel_two1(
    double *lhs_, double *lhsp_, double *lhsm_,
    double *rho_i, double *us, double *speed,
    int nx2, int ny2, int nz2,
    double c3c4, double dz4, double con43, double dz5,
    double c1c5, double dzmax, double dz1, double dttz2, double dttz1,
    double c2dttz1, double comz1, double comz4, double comz5, double comz6)
{
    int m;
    double ru1, rhos1;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = 1;

    if (j <= ny2 && i <= nx2)
    {
        lhs_(i, j, k, 0) = 0.0;

        ru1 = c3c4 * rho_i(k - 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 1) = -dttz2 * us(k - 1, i, j) - dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 2) = 1.0 + c2dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k + 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 3) = dttz2 * us(k + 1, i, j) - dttz1 * rhos1;
        lhs_(i, j, k, 4) = 0.0;

        lhs_(i, j, k, 1) = lhs_(i, j, k, 1) - comz4;
        lhs_(i, j, k, 2) = lhs_(i, j, k, 2) + comz6;
        lhs_(i, j, k, 3) = lhs_(i, j, k, 3) - comz4;
        lhs_(i, j, k, 4) = lhs_(i, j, k, 4) + comz1;

        lhsp_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsp_(i, j, k, 1) = lhs_(i, j, k, 1) - dttz2 * speed(k - 1, i, j);
        lhsp_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsp_(i, j, k, 3) = lhs_(i, j, k, 3) + dttz2 * speed(k + 1, i, j);
        lhsp_(i, j, k, 4) = lhs_(i, j, k, 4);
        lhsm_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsm_(i, j, k, 1) = lhs_(i, j, k, 1) + dttz2 * speed(k - 1, i, j);
        lhsm_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsm_(i, j, k, 3) = lhs_(i, j, k, 3) - dttz2 * speed(k + 1, i, j);
        lhsm_(i, j, k, 4) = lhs_(i, j, k, 4);
    }
};

__global__ void solve_kernel_two2(
    double *lhs_, double *lhsp_, double *lhsm_,
    double *rho_i, double *us, double *speed,
    int nx2, int ny2, int nz2,
    double c3c4, double dz4, double con43, double dz5,
    double c1c5, double dzmax, double dz1, double dttz2, double dttz1,
    double c2dttz1, double comz1, double comz4, double comz5, double comz6)
{
    int m;
    double ru1, rhos1;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = 2;

    if (j <= ny2 && i <= nx2)
    {
        lhs_(i, j, k, 0) = 0.0;

        ru1 = c3c4 * rho_i(k - 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 1) = -dttz2 * us(k - 1, i, j) - dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 2) = 1.0 + c2dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k + 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 3) = dttz2 * us(k + 1, i, j) - dttz1 * rhos1;
        lhs_(i, j, k, 4) = 0.0;

        lhs_(i, j, k, 1) = lhs_(i, j, k, 1) - comz4;
        lhs_(i, j, k, 2) = lhs_(i, j, k, 2) + comz5;
        lhs_(i, j, k, 3) = lhs_(i, j, k, 3) - comz4;
        lhs_(i, j, k, 4) = lhs_(i, j, k, 4) + comz1;

        lhsp_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsp_(i, j, k, 1) = lhs_(i, j, k, 1) - dttz2 * speed(k - 1, i, j);
        lhsp_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsp_(i, j, k, 3) = lhs_(i, j, k, 3) + dttz2 * speed(k + 1, i, j);
        lhsp_(i, j, k, 4) = lhs_(i, j, k, 4);
        lhsm_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsm_(i, j, k, 1) = lhs_(i, j, k, 1) + dttz2 * speed(k - 1, i, j);
        lhsm_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsm_(i, j, k, 3) = lhs_(i, j, k, 3) - dttz2 * speed(k + 1, i, j);
        lhsm_(i, j, k, 4) = lhs_(i, j, k, 4);
    }
};

__global__ void solve_kernel_two_nz2(
    double *lhs_, double *lhsp_, double *lhsm_,
    double *rho_i, double *us, double *speed,
    int nx2, int ny2, int nz2,
    double c3c4, double dz4, double con43, double dz5,
    double c1c5, double dzmax, double dz1, double dttz2, double dttz1,
    double c2dttz1, double comz1, double comz4, double comz5, double comz6)
{
    int m;
    double ru1, rhos1;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = nz - 2;

    if (j <= ny2 && i <= nx2)
    {
        lhs_(i, j, k, 0) = 0.0;

        ru1 = c3c4 * rho_i(k - 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 1) = -dttz2 * us(k - 1, i, j) - dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 2) = 1.0 + c2dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k + 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 3) = dttz2 * us(k + 1, i, j) - dttz1 * rhos1;
        lhs_(i, j, k, 4) = 0.0;

        lhs_(i, j, k, 0) = lhs_(i, j, k, 0) + comz1;
        lhs_(i, j, k, 1) = lhs_(i, j, k, 1) - comz4;
        lhs_(i, j, k, 2) = lhs_(i, j, k, 2) + comz5;

        lhsp_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsp_(i, j, k, 1) = lhs_(i, j, k, 1) - dttz2 * speed(k - 1, i, j);
        lhsp_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsp_(i, j, k, 3) = lhs_(i, j, k, 3) + dttz2 * speed(k + 1, i, j);
        lhsp_(i, j, k, 4) = lhs_(i, j, k, 4);
        lhsm_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsm_(i, j, k, 1) = lhs_(i, j, k, 1) + dttz2 * speed(k - 1, i, j);
        lhsm_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsm_(i, j, k, 3) = lhs_(i, j, k, 3) - dttz2 * speed(k + 1, i, j);
        lhsm_(i, j, k, 4) = lhs_(i, j, k, 4);
    }
};

__global__ void solve_kernel_two_nz3(
    double *lhs_, double *lhsp_, double *lhsm_,
    double *rho_i, double *us, double *speed,
    int nx2, int ny2, int nz2,
    double c3c4, double dz4, double con43, double dz5,
    double c1c5, double dzmax, double dz1, double dttz2, double dttz1,
    double c2dttz1, double comz1, double comz4, double comz5, double comz6)
{
    int m;
    double ru1, rhos1;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = nz - 3;

    if (j <= ny2 && i <= nx2)
    {
        lhs_(i, j, k, 0) = 0.0;

        ru1 = c3c4 * rho_i(k - 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 1) = -dttz2 * us(k - 1, i, j) - dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 2) = 1.0 + c2dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k + 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 3) = dttz2 * us(k + 1, i, j) - dttz1 * rhos1;
        lhs_(i, j, k, 4) = 0.0;

        lhs_(i, j, k, 0) = lhs_(i, j, k, 0) + comz1;
        lhs_(i, j, k, 1) = lhs_(i, j, k, 1) - comz4;
        lhs_(i, j, k, 2) = lhs_(i, j, k, 2) + comz5;
        lhs_(i, j, k, 3) = lhs_(i, j, k, 3) - comz4;

        lhsp_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsp_(i, j, k, 1) = lhs_(i, j, k, 1) - dttz2 * speed(k - 1, i, j);
        lhsp_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsp_(i, j, k, 3) = lhs_(i, j, k, 3) + dttz2 * speed(k + 1, i, j);
        lhsp_(i, j, k, 4) = lhs_(i, j, k, 4);
        lhsm_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsm_(i, j, k, 1) = lhs_(i, j, k, 1) + dttz2 * speed(k - 1, i, j);
        lhsm_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsm_(i, j, k, 3) = lhs_(i, j, k, 3) - dttz2 * speed(k + 1, i, j);
        lhsm_(i, j, k, 4) = lhs_(i, j, k, 4);
    }
};

__global__ void solve_kernel_two(
    double *lhs_, double *lhsp_, double *lhsm_,
    double *rho_i, double *us, double *speed,
    int nx2, int ny2, int nz2,
    double c3c4, double dz4, double con43, double dz5,
    double c1c5, double dzmax, double dz1, double dttz2, double dttz1,
    double c2dttz1, double comz1, double comz4, double comz5, double comz6)
{
    int m;
    double ru1, rhos1;

    int i = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int j = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 3;

    if (j <= ny2 && i <= nx2 && (k <= nz2 - 2))
    {
        lhs_(i, j, k, 0) = 0.0;

        ru1 = c3c4 * rho_i(k - 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 1) = -dttz2 * us(k - 1, i, j) - dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 2) = 1.0 + c2dttz1 * rhos1;

        ru1 = c3c4 * rho_i(k + 1, i, j);
        rhos1 = fmax(fmax(dz4 + con43 * ru1, dz5 + c1c5 * ru1), fmax(dzmax + ru1, dz1));
        lhs_(i, j, k, 3) = dttz2 * us(k + 1, i, j) - dttz1 * rhos1;
        lhs_(i, j, k, 4) = 0.0;

        lhs_(i, j, k, 0) = lhs_(i, j, k, 0) + comz1;

        lhs_(i, j, k, 1) = lhs_(i, j, k, 1) - comz4;
        lhs_(i, j, k, 2) = lhs_(i, j, k, 2) + comz6;
        lhs_(i, j, k, 3) = lhs_(i, j, k, 3) - comz4;
        lhs_(i, j, k, 4) = lhs_(i, j, k, 4) + comz1;

        lhsp_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsp_(i, j, k, 1) = lhs_(i, j, k, 1) - dttz2 * speed(k - 1, i, j);
        lhsp_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsp_(i, j, k, 3) = lhs_(i, j, k, 3) + dttz2 * speed(k + 1, i, j);
        lhsp_(i, j, k, 4) = lhs_(i, j, k, 4);
        lhsm_(i, j, k, 0) = lhs_(i, j, k, 0);
        lhsm_(i, j, k, 1) = lhs_(i, j, k, 1) + dttz2 * speed(k - 1, i, j);
        lhsm_(i, j, k, 2) = lhs_(i, j, k, 2);
        lhsm_(i, j, k, 3) = lhs_(i, j, k, 3) - dttz2 * speed(k + 1, i, j);
        lhsm_(i, j, k, 4) = lhs_(i, j, k, 4);
    }
};